#include "hip/hip_runtime.h"

#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <hip/hip_vector_types.h>
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))
#include "camera.h"
#include "kernel.h"
#include "loadData.h"
#include "structs.h"
#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <tira/image.h>
#include <tira/parser.h>

__constant__ float intersect_distance_maximum =
    std::numeric_limits<float>::max();
__constant__ int nTriangles_gpu;
__constant__ int nSpheres_gpu;
__constant__ int nPlanes_gpu;
__constant__ int nLights_gpu;
__constant__ int y_res;
__constant__ int x_res;

static void HandleError(hipError_t err, const char *file, int line) {
  if (err != hipSuccess)
    std::cout << hipGetErrorString(err) << " in " << file << " at line "
              << line << "\n";
}

int main(int argc, char *argv[]) {
  char *file = argv[1];

  tira::parser scene_file(file);
  int nLights = scene_file.count("light");
  struct pointSource *lights =
      (struct pointSource *)malloc(nLights * sizeof(struct pointSource));
  setLights(scene_file, lights);
  struct pointSource *lights_gpu = NULL;

  HANDLE_ERROR(hipMalloc(&lights_gpu, sizeof(struct pointSource)));
  HANDLE_ERROR(hipMemcpy(lights_gpu, lights,
                          nLights * sizeof(struct pointSource),
                          hipMemcpyHostToDevice));
  int *resolution = setImage(scene_file);
  int nSpheres = scene_file.count("sphere");
  int nPlanes = scene_file.count("plane");
  int nMeshes = scene_file.count("mesh");
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(&nSpheres_gpu), &nSpheres, 1 * sizeof(int)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(&nPlanes_gpu), &nPlanes, 1 * sizeof(int)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(&nLights_gpu), &nPlanes, 1 * sizeof(int)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(&x_res), &resolution[0], 1 * sizeof(int)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(&y_res), &resolution[1], 1 * sizeof(int)));

  // cuda setup
  int *device_count;
  HANDLE_ERROR(hipGetDeviceCount(device_count));
  if (*device_count == 0) {
    std::cout << "There are no available CUDA devices\n";
    return 0;
  } else
    std::cout << "There are " << *device_count << "CUDA devices\n";

  int *cuda_device;
  HANDLE_ERROR(hipGetDevice(cuda_device));
  hipDeviceProp_t *properties;
  HANDLE_ERROR(hipGetDeviceProperties(properties, *cuda_device));

  int nTriangles = 0;
  struct triangle *triangles_gpu = NULL;
  struct triangle *triangles = NULL;
  if (nMeshes > 0) {
    tira::parser mesh_file("./scenes/" +
                           scene_file.get<std::string>("mesh", 0));
    nTriangles = mesh_file.count("f");
    HANDLE_ERROR(
        hipMemcpyToSymbol(HIP_SYMBOL(&nTriangles_gpu), &nTriangles, 1 * sizeof(int)));
    triangles = (struct triangle *)malloc(nTriangles * sizeof(struct triangle));
    loadMesh(mesh_file, triangles, nTriangles);
    HANDLE_ERROR(
        hipMalloc(&triangles_gpu, nTriangles * sizeof(struct triangle)));
    HANDLE_ERROR(hipMemcpy(triangles_gpu, triangles,
                            (size_t)nTriangles * sizeof(struct triangle),
                            hipMemcpyHostToDevice));
  }

  struct sphere *spheres = NULL;
  struct sphere *spheres_gpu = NULL;
  if (nSpheres > 0) {
    spheres = (struct sphere *)malloc(nSpheres * sizeof(struct sphere));
    loadSpheres(scene_file, spheres, nSpheres);
    HANDLE_ERROR(hipMalloc(&spheres_gpu, nSpheres * sizeof(struct sphere)));
    HANDLE_ERROR(hipMemcpy(spheres_gpu, spheres,
                            (size_t)nSpheres * sizeof(struct sphere),
                            hipMemcpyHostToDevice));
  }

  struct plane *planes = NULL;
  struct plane *planes_gpu = NULL;
  if (nPlanes > 0) {
    planes = (struct plane *)malloc(nPlanes * sizeof(struct plane));
    loadPlanes(scene_file, planes, nPlanes);
    HANDLE_ERROR(hipMalloc(&planes_gpu, nPlanes * sizeof(struct plane)));
    HANDLE_ERROR(hipMemcpy(planes_gpu, planes,
                            (size_t)nPlanes * sizeof(struct plane),
                            hipMemcpyHostToDevice));
  }

  camera cam = setCamera(scene_file);
  struct camera2 *cam2;
  struct camera2 *cam2_gpu;
  cam2->_up = cam._up;
  cam2->_fov = cam._fov;
  cam2->_view = cam._view;
  cam2->_position = cam._position;
  cam2->_imagedist = cam._imagedist;
  cam2->_focus = cam._focus;
  HANDLE_ERROR(hipMalloc(&cam2, nPlanes * sizeof(struct camera2)));
  HANDLE_ERROR(hipMemcpy(cam2_gpu, cam2, 1 * sizeof(struct camera2),
                          hipMemcpyHostToDevice));
  glm::vec3 background_color = setBackground(scene_file);

  auto start = std::chrono::high_resolution_clock::now();

  tira::image<unsigned char> I(resolution[0], resolution[1], 3);

  unsigned char *image_data =
      new unsigned char[resolution[0] * resolution[1] * 3]();
  unsigned char *image_data_gpu = NULL;

  HANDLE_ERROR(hipMalloc(&image_data_gpu, resolution[0] * resolution[1] * 3));

  int block_side_length = std::sqrt(properties->maxThreadsPerBlock);
  int block_side_length2 = block_side_length;
  int nBlocks_y = resolution[1] / block_side_length + 1;
  int nBlocks_x = resolution[0] / block_side_length + 1;

  kernelPerPixel<<<dim3(nBlocks_x, nBlocks_y),
                   dim3(block_side_length, block_side_length2)>>>(
      image_data_gpu, cam2_gpu, planes_gpu, triangles_gpu, spheres_gpu,
      lights_gpu, y_res, x_res, intersect_distance_maximum, nSpheres, nPlanes,
      nTriangles, nLights);

  hipDeviceSynchronize();

  HANDLE_ERROR(
      hipMemcpy(image_data_gpu, image_data,
                 resolution[0] * resolution[1] * 3 * sizeof(unsigned char),
                 hipMemcpyDeviceToHost));

  for (int y = 0; y < resolution[1]; y++) {
    int y_i = resolution[1] - y - 1;
    for (int x = 0; x < resolution[0]; x++) {
      I(x, y, 0) = image_data[(y_i * resolution[0] + x) * 3 + 0];
      I(x, y, 1) = image_data[(y_i * resolution[0] + x) * 3 + 1];
      I(x, y, 2) = image_data[(y_i * resolution[0] + x) * 3 + 2];
    }
  }

  I.save("output.bmp");

  delete[] image_data;
  hipFree(image_data_gpu);
  hipFree(cam2_gpu);
  hipFree(planes_gpu);
  hipFree(spheres_gpu);
  hipFree(triangles_gpu);
  hipFree(lights_gpu);

  auto endTime = std::chrono::high_resolution_clock::now();
  auto duration =
      std::chrono::duration_cast<std::chrono::milliseconds>(endTime - start)
          .count();
  std::cout << "---Timing---\n"
            << "Total Trace Time (not including loading file): "
            << ((double)duration / (1'000.0));

  return 0;
}
