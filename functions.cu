#include "hip/hip_runtime.h"
#include "kernel.h"
#include "structs.h"
#include <__clang_cuda_builtin_vars.h>
#include <algorithm>
#include <hip/driver_types.h>
#include <glm/glm.hpp>

__device__ glm::vec3 rayAt(const struct ray &r, float t) {
  return r.orig + t * r.dir;
};

__device__ bool triangleHit(triangle tr, const ray &r, float max_dist,
                            hitRecord &rec) {
  // std::cout << "Ray:\nDirection -> x " << r.direction().x << ", y " <<
  // r.direction().y << ", z " << r.direction().y;
  glm::vec3 l = tr.m_centroid - r.orig;
  float l_squared = glm::dot(l, l);
  float radius_squared = tr.m_radius * tr.m_radius;

  // Test if the sphere is behind the camera
  float s = glm::dot(l, r.dir);
  if (s < 0) {
    // std::cout << "The bounding sphere is behind the camera\n";
    return false;
  }

  // Test if the ray intersects the sphere
  float m_squared = l_squared - (s * s);
  if (m_squared > radius_squared) {
    return false;
  }

  // Begin Plane Intersect test
  float denom = glm::dot(tr.m_normal, r.dir);

  if (fabs(denom) < 1e-6) {
    // std::cout << "The ray is parallel to the plane\n";
    return false;
  }

  float t = glm::dot(tr.m_normal, (tr.m_v[0] - r.orig)) / denom;
  if (t < 0 || t > max_dist) {
    // std::cout << "The distance to the point is out of range\n";
    return false;
  }

  // Find intersect point on plane and test if point is inside triangle
  glm::vec3 p = rayAt(r, t);

  // To the left of first line
  glm::vec3 pa = p - tr.m_v[0];
  float cross1 = glm::dot(glm::cross(tr.m_v0v1, pa), tr.m_normal);
  if (cross1 < 0) {
    // std::cout << "Not Left of first line\n";
    return false;
  }

  // To the left of second line
  glm::vec3 pb = p - tr.m_v[1];
  float cross2 = glm::dot(glm::cross(tr.m_v1v2, pb), tr.m_normal);
  if (cross2 < 0) {
    // std::cout << "Not Left of second line\n";
    return false;
  }

  // To the left of third line
  glm::vec3 pc = p - tr.m_v[2];
  float cross3 = glm::dot(glm::cross(tr.m_v2v0, pc), tr.m_normal);
  if (cross3 < 0) {
    // std::cout << "Not Left of third line\n";
    return false;
  }

  rec.t = t;
  rec.p = p;
  rec.normal = tr.m_normal;
  rec.color = tr.m_color;

  // std::cout << "Intersect detected\n";

  return true;
}
__device__ bool planeHit(plane p, const ray &r, float max_dist,
                         hitRecord &rec) {
  float denom = glm::dot(p.m_normal, r.dir);

  if (std::fabs(denom) < 1e-6)
    return false;

  float t = glm::dot(p.m_normal, (p.m_point - r.orig)) / denom;
  if (t < 0 || t > max_dist)
    return false;

  rec.t = t;
  rec.p = rayAt(r, t);
  rec.normal = p.m_normal;
  rec.color = p.m_color;
  return true;
}
__device__ bool sphereHit(const struct sphere &sph, const ray &r,
                          float max_dist, struct hitRecord &rec) {

  // Test if the camera is inside the sphere
  // std::cout << "Ray:\nDirection -> x " << r.direction().x << ", y " <<
  // r.direction().y << ", z " << r.direction().y;
  glm::vec3 l = sph.m_center - r.orig;
  float l_squared = glm::dot(l, l);
  float radius_squared = sph.m_radius * sph.m_radius;
  if (l_squared < radius_squared) {
    return false;
  }

  // Test if the sphere is behind the camera
  float s = glm::dot(l, r.dir);
  if (s < 0) {
    //      std::cout << "SPHERE BEHIND CAMERA\n";
    return false;
  }

  // Test if the ray intersects the sphere
  float m_squared = l_squared - (s * s);
  if (m_squared >
      radius_squared) { //            std::cout << "M is greater than R\n";
    return false;
  }

  // Return the position of the intersect and the normal of the sphere at that
  // point
  float q_squared = radius_squared - m_squared;
  float q = std::sqrt(q_squared);
  float t = s - q;
  if (t > max_dist) {
    // std::cout << "SPHERE DISTANCE TOO LARGE\n";
    return false;
  }
  rec.t = t;
  rec.p = rayAt(r, t);
  rec.normal = glm::normalize((rec.p - sph.m_center));
  rec.color = sph.m_color;

  return true;
}

__device__ bool closestHit(const ray &r, float max_val, hitRecord &rec,
                           const triangle *triangles, const sphere *spheres,
                           const plane *planes, int nSpheres, int nPlanes,
                           int nTriangles) {
  hitRecord temporary_record;
  bool hit_anything = false;
  float closest_so_far = max_val;
  int i = 0;

  for (size_t i = 0; i < nSpheres; i++) {
    if (sphereHit(spheres[i], r, closest_so_far, temporary_record)) {
      // std::cout << "an object was hit\n";
      hit_anything = true;
      closest_so_far = temporary_record.t;
      rec = temporary_record;
      rec.object_ID = i;
    }
  }
  for (size_t i = 0; i < nPlanes; i++) {
    if (planeHit(planes[i], r, closest_so_far, temporary_record)) {
      // std::cout << "an object was hit\n";
      hit_anything = true;
      closest_so_far = temporary_record.t;
      rec = temporary_record;
      rec.object_ID = i;
    }
  }
  for (size_t i = 0; i < nTriangles; i++) {
    if (triangleHit(triangles[i], r, closest_so_far, temporary_record)) {
      // std::cout << "an object was hit\n";
      hit_anything = true;
      closest_so_far = temporary_record.t;
      rec = temporary_record;
      rec.object_ID = i;
    }
  }
  return hit_anything;
}

__device__ bool lightHit(const ray &r, float max_val, hitRecord &current_point,
                         const triangle *triangles, const sphere *spheres,
                         const plane *planes, int nSpheres, int nPlanes,
                         int nTriangles) {
  hitRecord temporary_record;

  for (size_t i = 0; i < nSpheres; i++) {
    if (sphereHit(spheres[i], r, max_val, temporary_record) &&
        i != current_point.object_ID) {
      return true;
    }
  }
  for (size_t i = 0; i < nPlanes; i++) {
    if (planeHit(planes[i], r, max_val, temporary_record) &&
        i != current_point.object_ID) {
      return true;
    }
  }
  for (size_t i = 0; i < nTriangles; i++) {
    if (triangleHit(triangles[i], r, max_val, temporary_record) &&
        i != current_point.object_ID) {
      return true;
    }
  }
  return false;
}

__device__ ray generateRay(struct pointSource s, glm::vec3 point) {
  // no need to normalize because you wont need the distance from the origin to
  // the intersect for lighting intersections.
  ray r;
  r.dir = glm::normalize(s.pos - point);
  r.orig = point;
  return r;
}

__device__ glm::vec3
onHitShadow(const sphere *sp, const triangle *tr, const plane *pl,
            hitRecord &rec, float intersect_distance_maximum,
            int number_of_lights, const struct pointSource *lights,
            const int nSpheres, const int nTriangles, const int nPlanes) {
  glm::vec3 color_intensity{0.0, 0.0, 0.0};
  hitRecord temp_record;
  float intensity;
  glm::vec3 color;
  ray lighting_ray;

  for (int i = 0; i < number_of_lights; i++) {
    lighting_ray = generateRay(lights[i], rec.p);
    // std::cout << "LIGHT VECTOR: x " << lighting_ray.direction().x << ", y "
    // << lighting_ray.direction().y << ", z " << lighting_ray.direction().z <<
    // "\n"; std::cout << "NORMAL VECTOR: x " << rec.normal.x << ", y " <<
    // rec.normal.y << ", z " << rec.normal.z << "\n";
    intensity =
        std::max(glm::dot(lighting_ray.dir, rec.normal), static_cast<float>(0));
    // std::cout << "intensity: " << intensity << "\n";
    if (intensity > static_cast<float>(0)) {
      if (!lightHit(lighting_ray, intersect_distance_maximum, rec, tr, sp, pl,
                    nSpheres, nPlanes, nTriangles)) {
        color_intensity.r += intensity * lights[i].col.r;
        color_intensity.g += intensity * lights[i].col.g;
        color_intensity.b += intensity * lights[i].col.b;
      }
    }
  }
  color = {std::clamp(color_intensity.r, 0.0f, 1.0f) * rec.color.r,
           std::clamp(color_intensity.g, 0.0f, 1.0f) * rec.color.g,
           std::clamp(color_intensity.b, 0.0f, 1.0f) * rec.color.b};

  return color;
}

__device__ glm::vec3 cameraRay(const struct camera2 &cam, const float x,
                               const float y) {
  const glm::vec3 side = glm::cross(cam._view, cam._up);
  const glm::vec3 result = x * side + y * cam._up + cam._imagedist * cam._view;
  return glm::normalize(result);
}

__global__ void
kernelPerPixel(unsigned char *image_data, const struct camera2 *cam,
               const struct plane *pl, const struct triangle *tr,
               const struct sphere *sp, const struct pointSource *lights,
               const int y_res, const int x_res,
               float intersect_distance_maximum, int nSpheres, int nPlanes,
               int nTriangles, int nLights) {
  float x_pixel_position;
  float y_pixel_position;
  glm::vec3 color;
  hitRecord rec;

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < x_res && y < y_res) {

    x_pixel_position = (static_cast<float>(x) / (float)y_res - 0.5f);
    y_pixel_position = (static_cast<float>(y) / (float)x_res - 0.5f);
    struct ray r;
    r.orig = cam->_position;
    r.dir = glm::normalize(cameraRay(*cam, x_pixel_position, y_pixel_position));
    if (closestHit(r, intersect_distance_maximum, rec, tr, sp, pl, nSpheres,
                   nPlanes, nTriangles)) {
      color = onHitShadow(sp, tr, pl, rec, intersect_distance_maximum, nLights,
                          lights, nSpheres, nTriangles, nPlanes);
      image_data[3 * (y * y_res + x) + 0] =
          static_cast<unsigned char>(255 * color.r);
      image_data[3 * (y * y_res + x) + 1] =
          static_cast<unsigned char>(255 * color.g);
      image_data[3 * (y * y_res + x) + 2] =
          static_cast<unsigned char>(255 * color.b);
    } else {
      image_data[3 * (y * y_res + x) + 0] = 0;
      image_data[3 * (y * y_res + x) + 1] = 0;
      image_data[3 * (y * y_res + x) + 2] = 0;
    }
  }
}
